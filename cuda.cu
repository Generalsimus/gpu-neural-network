#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

// CUDA kernel to add two arrays
__global__ void addArrays(float *a, float *b, float *c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

typedef struct Inputs
{
    float *allocatedInputs;
    int count;
} Inputs;

typedef struct Connect
{
    float *widths;
    float *biases;
} Connect;

typedef struct Model
{
    Inputs *inputs;
    Connect *connects;
    int count;
} Model;

// typedef struct Connects
// {
//     Connect *Connects;
//     int count;
// } Connects;

float *AllocateGpuFloatArray(int size)
{
    float *input = (float *)malloc(size * sizeof(float));

    float *d_input;

    hipMalloc((void **)&d_input, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    free(input);

    return d_input;
}

// float GetWidthIndex(Elements model, int layerIndex, int inputIndex, int outputIndex)
// {

//     int layersCount = model.count;
//     float *inputSizes = model.elements;
//     for (int inputIndex = 0; inputIndex < layersCount; inputIndex++)
//     {
//         int inputSize = inputSizes[inputIndex];

//         printf("GetWidthIndex: %d\n", inputSize);
//     }
//     // Elements model = {
//     //     sizes,
//     //     sizeof(sizes) / sizeof(sizes[0]),
//     // };
//     float retur = 0;
//     return retur;
// }
// float **AllocateModelInputs(Model model)
// {
//     int connectsCount = model.count - 1;
//     float** inputs;
//     hipMalloc((void**)&inputs, connectsCount * sizeof(float*));
//     for (int connectionIndx = 0; connectionIndx < connectsCount; connectionIndx++)
//     {
// inputs[connectionIndx] =
//     }
// }

float *Forwards(Connect *LayerConnect, float *inputs, float *allocatedOutput, int inputSize, int outputSize)
{

    for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
    {
        float output = 0;

        for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
        {
            float input = inputs[inputIndex];
            output += input * (outputIndex * inputSize + inputIndex);
        };
        allocatedOutput[outputIndex] = output;
    }
}

float *Forwards(Model model, Connect *LayerConnects, float *input, float *output)
{

    int connectsCount = model.count - 1;
    for (int connectionIndx = 0; connectionIndx < connectsCount; connectionIndx++)
    {

        int inputSize = model.layersSizes[connectionIndx - 1];
        int outputSize = model.layersSizes[connectionIndx];

        float *currentOutput = (float *)malloc(outputSize * sizeof(float));

        Connect connect = LayerConnects[connectionIndx];

        float *currentWidths = connect.widths;
        float *currentBiases = connect.biases;

        // for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
        // {
        //     float outputForwardValue = 0;
        //     for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
        //     {
        //     }

        //     currentOutput[outputIndex] = outputForwardValue;
        // }

        // float *currentOutput = (float *)malloc(outputSize * sizeof(float));
    }

    // int layersCount = model.count;
    // float *inputSizes = model.elements;
    // int layerStartAt = 0;
    // int prevInputsSize = 0;
    // for (int layerIndex = 0; layerIndex < layersCount; layerIndex++)
    // {
    //     int inputSize = inputSizes[layerIndex];

    //     // if (layerIndex != 0)
    //     // {
    //     // for (int inputIndex = 0; inputIndex < inputSize; inputSize++)
    //     // {
    //     // }
    //     // }

    //     printf("Forwards: %d\n", inputSize);
    //     for (int inputIndex = 0; inputIndex < inputSize; inputSize++)
    //     {
    //     }

    //     layerStartAt = layerStartAt + (inputSize * prevInputsSize);
    //     prevInputsSize = inputSize;
    // }
}
// Elements GetForwardinput(Elements widths, Elements input, int layerStartAt, int inputSize, int OutputSize)
// {

//     float asssss[] = {3, 5, 2};
//     Elements aaa = {
//         asssss,
//         3,
//     };
//     return aaa;
// };

Model CreateModel(int layersSizes[], int count)
{
    Connect *LayerConnects = (Connect *)malloc((count - 1) * sizeof(Connect));
    Inputs *LayerInputs = (Inputs *)malloc(count * sizeof(Inputs));

    int prevLayerInputsSize = 0;
    for (int i = 0; i < count; i++)
    {
        int layerInputsSize = layersSizes[i];

        Inputs inputs = {
            AllocateGpuFloatArray(layerInputsSize),
            layerInputsSize,
        };

        LayerInputs[i] = inputs;
        if (i == 0)
        {
            Connect connect = {
                AllocateGpuFloatArray(layerInputsSize * prevLayerInputsSize),
                AllocateGpuFloatArray(layerInputsSize),
            };

            LayerConnects[i] = connect;
        };

        printf("Element %d: %d\n", i, layerInputsSize);
        prevLayerInputsSize = layerInputsSize;
    };
    Model model = {
        LayerInputs,
        LayerConnects,
        count,
    };
    return model;
}
int main()
{

    int sizes[] = {3, 5, 2};
    // Model model = {
    //     sizes,
    //     sizeof(sizes) / sizeof(sizes[0]),
    // };

    Connect *Connects = CreateModel(sizes, 3);

    float inputs[] = {3, 5, 2};
    float outputs[] = {3, 5};

    Forwards(model, Connects, inputs, outputs);

    // printf("Eleqweqwement: %d\n", sizes);
    ////
    // const int arraySize = 10;
    // const int arrayBytes = arraySize * sizeof(float);

    // // Input arrays and output array on the host (CPU)
    // float a[arraySize] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    // float b[arraySize] = {10.0, 9.0, 8.0, 7.0, 6.0, 5.0, 4.0, 3.0, 2.0, 1.0};
    // float c[arraySize] = {0};

    // // Device (GPU) pointers for arrays
    // float *dev_a, *dev_b, *dev_c;

    // // Allocate memory on the GPU
    // hipMalloc((void**)&dev_a, arrayBytes);
    // hipMalloc((void**)&dev_b, arrayBytes);
    // hipMalloc((void**)&dev_c, arrayBytes);

    // // Copy input arrays from host to device
    // hipMemcpy(dev_a, a, arrayBytes, hipMemcpyHostToDevice);
    // hipMemcpy(dev_b, b, arrayBytes, hipMemcpyHostToDevice);

    // // Launch kernel on the GPU
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    // addArrays<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, arraySize);

    // // Copy the result from device to host
    // hipMemcpy(c, dev_c, arrayBytes, hipMemcpyDeviceToHost);

    // // Free memory on the GPU
    // hipFree(dev_a);
    // hipFree(dev_b);
    // hipFree(dev_c);

    // // Print the result
    // for (int i = 0; i < arraySize; ++i) {
    //     std::cout << c[i] << " ";
    // }
    // std::cout << std::endl;

    return 0;
}