#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

// CUDA kernel to add two arrays
__global__ void addArrays(float *a, float *b, float *c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

typedef struct Model
{
    float *elements;
    int count;
} Model;

typedef struct Connect
{
     float *widths;
     float *biases;
} Connect;
// typedef struct Connects
// {
//     Connect *Connects;
//     int count;
// } Connects;

Elements AllocateGpuFloatArray(int size)
{
    float *input = (float *)malloc(size * sizeof(float));

    float *d_input;

    hipMalloc((void **)&d_input, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    free(input);
    // sssssssss213
    // hipFree(d_input);

    Elements widths = {
        d_input,
        size,
    };
    return widths;
}

// float GetWidthIndex(Elements model, int layerIndex, int inputIndex, int outputIndex)
// {

//     int layersCount = model.count;
//     float *inputSizes = model.elements;
//     for (int inputIndex = 0; inputIndex < layersCount; inputIndex++)
//     {
//         int inputSize = inputSizes[inputIndex];

//         printf("GetWidthIndex: %d\n", inputSize);
//     }
//     // Elements model = {
//     //     sizes,
//     //     sizeof(sizes) / sizeof(sizes[0]),
//     // };
//     float retur = 0;
//     return retur;
// }

Elements Forwards(Elements model, Connects LayerConnects, Elements input, Elements output)
{

    for (int connectionIndx = 0; connectionIndx < LayerConnects.count; connectionIndx++)
    {
        int inputSize = model.elements[connectionIndx - 1];
        int outputSize = model.elements[connectionIndx];

        float *currentOutput = (float *)malloc(outputSize * sizeof(float));

        Connect connect = LayerConnects.Connects[connectionIndx];

        Elements currentWidths = connect.widths;
        Elements currentBiases = connect.biases;

        // for (int connectionIndx = 0; connectionIndx < LayerConnects->count; connectionIndx++)
        // {
        // }

        // float *currentOutput = (float *)malloc(outputSize * sizeof(float));
    }

    // int layersCount = model.count;
    // float *inputSizes = model.elements;
    // int layerStartAt = 0;
    // int prevInputsSize = 0;
    // for (int layerIndex = 0; layerIndex < layersCount; layerIndex++)
    // {
    //     int inputSize = inputSizes[layerIndex];

    //     // if (layerIndex != 0)
    //     // {
    //     // for (int inputIndex = 0; inputIndex < inputSize; inputSize++)
    //     // {
    //     // }
    //     // }

    //     printf("Forwards: %d\n", inputSize);
    //     for (int inputIndex = 0; inputIndex < inputSize; inputSize++)
    //     {
    //     }

    //     layerStartAt = layerStartAt + (inputSize * prevInputsSize);
    //     prevInputsSize = inputSize;
    // }
}
// Elements GetForwardinput(Elements widths, Elements input, int layerStartAt, int inputSize, int OutputSize)
// {

//     float asssss[] = {3, 5, 2};
//     Elements aaa = {
//         asssss,
//         3,
//     };
//     return aaa;
// };

Connects CreateModel(Elements model)
{
    int layersCount = model.count;
    float *inputSizes = model.elements;

    Connect *LayerConnects = (Connect *)malloc(layersCount * sizeof(Connect));
    int prevLayerInputsSize = 0;

    for (int i = 0; i < layersCount; i++)
    {
        int layerInputsSize = inputSizes[i];
        if (i == 0)
        {
            Connect connect = {
                AllocateGpuFloatArray(layerInputsSize * prevLayerInputsSize),
                AllocateGpuFloatArray(layerInputsSize),
            };
            LayerConnects[i] = connect;

            // LayerConnects[i] = CreateWidth(layerInputsSize * prevLayerInputsSize);
        };

        printf("Element %d: %d\n", i, layerInputsSize);
        prevLayerInputsSize = layerInputsSize;
    };

    Connects Connects = {
        LayerConnects,
        layersCount,
    };

    return Connects;
}
int main()
{

    float sizes[] = {3, 5, 2};
    Elements model = {
        sizes,
        sizeof(sizes) / sizeof(sizes[0]),
    };

    Connects Connects = CreateModel(model);

    float inputs[] = {3, 5, 2};
    Elements input = {
        inputs,
        3,
    };
    float outputs[] = {3, 5};
    Elements output = {
        outputs,
        2,

    };
    Forwards(model, Connects, input, output);

    // printf("Eleqweqwement: %d\n", sizes);
    ////
    // const int arraySize = 10;
    // const int arrayBytes = arraySize * sizeof(float);

    // // Input arrays and output array on the host (CPU)
    // float a[arraySize] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    // float b[arraySize] = {10.0, 9.0, 8.0, 7.0, 6.0, 5.0, 4.0, 3.0, 2.0, 1.0};
    // float c[arraySize] = {0};

    // // Device (GPU) pointers for arrays
    // float *dev_a, *dev_b, *dev_c;

    // // Allocate memory on the GPU
    // hipMalloc((void**)&dev_a, arrayBytes);
    // hipMalloc((void**)&dev_b, arrayBytes);
    // hipMalloc((void**)&dev_c, arrayBytes);

    // // Copy input arrays from host to device
    // hipMemcpy(dev_a, a, arrayBytes, hipMemcpyHostToDevice);
    // hipMemcpy(dev_b, b, arrayBytes, hipMemcpyHostToDevice);

    // // Launch kernel on the GPU
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    // addArrays<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, arraySize);

    // // Copy the result from device to host
    // hipMemcpy(c, dev_c, arrayBytes, hipMemcpyDeviceToHost);

    // // Free memory on the GPU
    // hipFree(dev_a);
    // hipFree(dev_b);
    // hipFree(dev_c);

    // // Print the result
    // for (int i = 0; i < arraySize; ++i) {
    //     std::cout << c[i] << " ";
    // }
    // std::cout << std::endl;

    return 0;
}