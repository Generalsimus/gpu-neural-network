#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

 

 
__global__ void ForwardKernel(float* inputs, int inputSize, float* output, int outputSize, float* widths)
{
    unsigned int widthIndex = (blockDim.x * blockIdx.x) + threadIdx.x;

    unsigned int inputIndex = widthIndex / outputSize;
    unsigned int outputIndex = widthIndex - (inputIndex * outputSize);


    output[outputIndex] += inputs[inputIndex] * widths[widthIndex];

   printf("widthIndex: %d \n", widthIndex);
   // printf("inputSize: %d \n", inputSize);
    //printf("inputIndex: %d, outputIndex: %d\n", inputIndex, outputIndex);
}
__global__ void SigmoidKernel(float* output, float* biases)
{
    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;



    output[outputIndex] = 1.0f / (1.0f + expf(-(output[outputIndex] + biases[outputIndex])));

    // printf("outputIndex: %d \n", outputIndex);
    //printf("output: %.2f\n", output[outputIndex]);
}
__global__ void JustForwardKernel(float* inputs, int inputSize, float* outputs, int outputSize, float* widths, float* biases)
{


    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;


    float output = 0;


    for (int inputIndex = 0; inputIndex < inputSize; ++inputIndex) {
        float width = widths[inputIndex + (outputIndex * inputSize)];
        output += width + inputs[inputIndex];
    }



    outputs[outputIndex] = output;
    // printf("outputIndex: %d \n", outputIndex);
    //printf("output: %.2f\n", outputs[outputIndex]);
};

 




 


int main()
{ 

    Channel chan = {};
    AddOutputInput(&chan, 5);
    AddOutputInput(&chan, 3); 


    float inputs[5] = {1,3,4,2,7};   
   // Inputs* forwardInputs = FloatToInputs(inputs, 5);
    Inputs forwardIn = FloatToInputs(inputs, 5);
    Inputs* forwardInputs = &forwardIn;

    MakeFillAllocatedOutputs(&chan, 0);
     
     
    ForWards(&chan, forwardInputs);

    hipError_t cudaStatus;

    LogInput(forwardInputs);











    cudaStatus = hipGetLastError();
    printf("ERROR: %d\n", cudaStatus != hipSuccess);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaStatus));
        // Handle or report the error appropriately
    }
   
    return 0;

 /*   int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);



    int inputSize = INT_MAX;
    int outputSize = INT_MAX;*/

    

    //int miniBatchSize = 111;
    //// Set the dimensions of the CUDA grid and blocks
    //hipDeviceProp_t prop;
    //hipGetDeviceProperties(&prop, 0);  // Assuming device 0
    //int maxThreadsXx = prop.maxThreadsDim[0];
    //int blockDimX = maxThreadsXx;
    ////int numBlocksX = (miniBatchSize + blockDimX - 1) / blockDimX;

    //int ic = inputSize * outputSize;

    //int threadsPerBlockX = std::min(ic, prop.maxThreadsPerBlock);
    //int numBlocksX = (ic + threadsPerBlockX - 1) / threadsPerBlockX;

    //// Calculate the total number of threads
    //int numThreads = numBlocksX * threadsPerBlockX;

    //printf("blockDimX: %d \n", blockDimX);
    //printf("numBlocksX: %d \n", numBlocksX);
    //dim3 blockSize(256,100);
    //printf("blockSizeX: %d \n", blockSize.x);
    //printf("blockSizeY: %d \n", blockSize.y);
    //printf("blockSizeZ: %d \n", blockSize.z);
    //
    //int *siz = CalctThreadsAndBlocks(inputSize * outputSize);


    /*dim3 blocksPerGrid(inputSize, 1, 1);
    dim3 threadsPerBlock(outputSize, 1, 1);*/
    //dim3 threadsPerBlock(inputSize, outputSize);
    //dim3 blocksPerGrid(1, 1);
    //int maxThreads = deviceProp.maxThreadsPerBlock / 2;

    //if ((inputSize * outputSize) > maxThreads) {
    //    threadsPerBlock.x = maxThreads;
    //    threadsPerBlock.y = maxThreads;
    //    blocksPerGrid.x = ceil(double(inputSize) / double(threadsPerBlock.x));
    //    blocksPerGrid.y = ceil(double(outputSize) / double(threadsPerBlock.y));
    //}

    ////
    //printf("B %d \n", ceil(inputSize/ deviceProp.maxThreadsPerBlock));
    ////printf("T %d \n", min(inputSize, outputSize));

    //printf("RUN %d \n", (inputSize * outputSize));
    //printf("threadsPerBlock: %d \n", threadsPerBlock.x);
    //printf("threadsPerBlock: %d \n", threadsPerBlock.y);
    //printf("blocksPerGrid: %d \n", blocksPerGrid.x);
    //printf("BLOCK_SIZE: %d \n", BLOCK_SIZE);
    /*dim3 blocksPerGrid(2);
    dim3 threadsPerBlock(1024, 1024);*/ 

    
    // const int inputsize = 200000;
    // const int outputsize = 30000;


    // float* input, *output, *widts, *biases;
    // hipMalloc((void**)&input, inputsize * sizeof(float));
    // hipMalloc((void**)&output, outputsize * sizeof(float));
    // hipMalloc((void**)&widts, inputsize* outputsize * sizeof(float));
    // hipMalloc((void**)&biases, outputsize * sizeof(float));

    // ////////////////////////////////////////////////////
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // // Start recording the execution time
    // hipEventRecord(start);
    // ////////////////////////////////////////////////////

    // //(inputsize * outputsize)

    // //ForwardKernel<<<(2147483647 * 50009001), 1 >> >(input, inputsize, output, outputsize, widts);
    // SigmoidKernel<<<1, outputsize>>>(output, biases);
    // ////////////////////////////////////////////////////
    //  // Stop recording the execution time
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);

    // // Calculate the elapsed time
    // float milliseconds = 0.0f;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // // Print the kernel execution time
    // std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // // Destroy CUDA events
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    // //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // hipEvent_t start2, stop2;
    // hipEventCreate(&start2);
    // hipEventCreate(&stop2);

    // // Start recording the execution time
    // hipEventRecord(start2);
    // ////////////////////////////////////////////////////

    // JustForwardKernel<<<1, outputsize>>>(input, inputsize, output, outputsize, widts, biases);
    // ////////////////////////////////////////////////////
    //  // Stop recording the execution time
    // hipEventRecord(stop2);
    // hipEventSynchronize(stop2);

    // // Calculate the elapsed time
    // float milliseconds2 = 0.0f;
    // hipEventElapsedTime(&milliseconds2, start2, stop2);

    // // Print the kernel execution time
    // std::cout << "Kernel execution time22: " << milliseconds2 << " ms" << std::endl;

    // // Destroy CUDA events
    // hipEventDestroy(start2);
    // hipEventDestroy(stop2);

    // //////////////////////////////////////////////////// 


    //  


    // int device;
    // hipGetDevice(&device);

    // hipDeviceProp_t devicePropp;
    // hipGetDeviceProperties(&devicePropp, device);

    // // Get the maximum number of threads per block
    // int maxThreadsPerBlock = devicePropp.maxThreadsPerBlock;

    // // Get the maximum dimensions of the grid
    // //dim3 maxGridSize = devicePropp.maxGridSize;


    // // Get the maximum dimensions of the grid
    // int maxBlocksPerMultiprocessor;
    // hipDeviceGetAttribute(&maxBlocksPerMultiprocessor, hipDeviceAttributeMaxBlocksPerMultiprocessor, device);

    //  
    //// printf("maxBlocksPerGrid123: %d\n", maxBlocksPerMultiprocessor);

    // //myKernel << <maxBlocksPerGrid, maxThreadsPerBlock >> > ();

    // int deviceCount;
    // hipGetDeviceCount(&deviceCount);

    // if (deviceCount == 0) {
    //     std::cerr << "No CUDA devices found" << std::endl;
    //     return 1;
    // }

    // for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
    //     hipDeviceProp_t deviceProp;
    //     hipGetDeviceProperties(&deviceProp, deviceId);

    //     std::cout << "Device ID: " << deviceId << std::endl;
    //     std::cout << "Device Name: " << deviceProp.name << std::endl;
    //     std::cout << "Max Blocks per SM: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
    //     std::cout << "Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
    //     std::cout << "Max Threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
    //     std::cout << "Max Grid Size (x, y, z): " << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1]
    //         << ", " << deviceProp.maxGridSize[2] << std::endl;
    //     std::cout << std::endl;
    //}
    //return 0;
}