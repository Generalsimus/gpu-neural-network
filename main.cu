#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

 

 
//__global__ void ForwardKernel(float* inputs, int inputSize, float* output, int outputSize, float* widths)
//{
//    unsigned int widthIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//    unsigned int inputIndex = widthIndex / outputSize;
//    unsigned int outputIndex = widthIndex - (inputIndex * outputSize);
//
//
//    output[outputIndex] += inputs[inputIndex] * widths[widthIndex];
//
//   printf("widthIndex: %d \n", widthIndex);
//   // printf("inputSize: %d \n", inputSize);
//    //printf("inputIndex: %d, outputIndex: %d\n", inputIndex, outputIndex);
//}
//__global__ void SigmoidKernel(float* output, float* biases)
//{
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//
//    output[outputIndex] = 1.0f / (1.0f + expf(-(output[outputIndex] + biases[outputIndex])));
//
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", output[outputIndex]);
//}
//__global__ void JustForwardKernel(float* inputs, int inputSize, float* outputs, int outputSize, float* widths, float* biases)
//{
//
//
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//    float output = 0;
//
//
//    for (int inputIndex = 0; inputIndex < inputSize; ++inputIndex) {
//        float width = widths[inputIndex + (outputIndex * inputSize)];
//        output += width + inputs[inputIndex];
//    }
//
//
//
//    outputs[outputIndex] = output;
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", outputs[outputIndex]);
//};

 
 

 
//unsigned

int main()
{

    float floatmin = FLT_MIN;

    printf("Minimal float value: %.100f \n", FLT_MIN);
    printf("Size of float variable: %zu bytes\n", sizeof(FLT_MIN));

    Channel chan = {};

    AddOutputInput(&chan, 5);
    AddOutputInput(&chan, 100);
    AddOutputInput(&chan, 3);


    float inputs[5] = { 1,3,4,2,7 };
    float inputs2[5] = { 3,3,5,8,7 };
   // Inputs* forwardInputs = FloatToInputs(inputs, 5);


    float* inputsNormalizedDeltas = NormalizeDeltas(inputs, 5);
    float* inputsNormalizedDeltas2 = NormalizeDeltas(inputs2, 5);

    Inputs forwardIn = FloatToInputs(inputsNormalizedDeltas, 5);
    Inputs forwardIn2 = FloatToInputs(inputsNormalizedDeltas2, 5);


      
    LogInput(&forwardIn);
    LogInput(&forwardIn2); 


    ////////////////////////////////////
    float trainDesiredOutputs[3] = { 0,0,1 };
    float trainDesiredOutputs2[3] = { 0,1,0 };

    Inputs trainDesiredOutputsForwardIn = FloatToInputs(trainDesiredOutputs, 3); 

    Inputs trainDesiredOutputsForwardIn2 = FloatToInputs(trainDesiredOutputs2, 3);
    ////////////////////////////////////

    for (int i = 0; i < 5000; i++) {
        MakeFillAllocatedOutputs(&chan, 0);
        Train(&chan, &forwardIn, &trainDesiredOutputsForwardIn, 0.2);
       // printf("INDDTRდ: %d \n", i);/*
        MakeFillAllocatedOutputs(&chan, 0);
        Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 0.2); 
    };

    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult3 = ForWards(&chan, &forwardIn);
    LogInput(&forwardResult3);


    ////////////////////////////////////////////
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);
    LogInput(&forwardResult4);



    hipError_t cudaStatus = hipGetLastError();

    printf("ERROR: %d\n", cudaStatus != hipSuccess);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaStatus));
        // Handle or report the error appropriately
    }
   
    return 0;


}