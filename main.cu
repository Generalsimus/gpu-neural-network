#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include "skia.cu"
#include <windows.h>


 

 
//__global__ void ForwardKernel(float* inputs, int inputSize, float* output, int outputSize, float* widths)
//{
//    unsigned int widthIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//    unsigned int inputIndex = widthIndex / outputSize;
//    unsigned int outputIndex = widthIndex - (inputIndex * outputSize);
//
//
//    output[outputIndex] += inputs[inputIndex] * widths[widthIndex];
//
//   printf("widthIndex: %d \n", widthIndex);
//   // printf("inputSize: %d \n", inputSize);
//    //printf("inputIndex: %d, outputIndex: %d\n", inputIndex, outputIndex);
//}
//__global__ void SigmoidKernel(float* output, float* biases)
//{
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//
//    output[outputIndex] = 1.0f / (1.0f + expf(-(output[outputIndex] + biases[outputIndex])));
//
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", output[outputIndex]);
//}
//__global__ void JustForwardKernel(float* inputs, int inputSize, float* outputs, int outputSize, float* widths, float* biases)
//{
//
//
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//    float output = 0;
//
//
//    for (int inputIndex = 0; inputIndex < inputSize; ++inputIndex) {
//        float width = widths[inputIndex + (outputIndex * inputSize)];
//        output += width + inputs[inputIndex];
//    }
//
//
//
//    outputs[outputIndex] = output;
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", outputs[outputIndex]);
//};

LRESULT CALLBACK WndProc(HWND hwnd, UINT msg, WPARAM wParam, LPARAM lParam) {
    switch (msg) {
        case WM_PAINT: {
            PAINTSTRUCT ps;
            HDC hdc = BeginPaint(hwnd, &ps);

            // Create Skia Surface and Canvas
            sk_sp<SkSurface> surface = SkSurface::MakeFromHWND(hwnd);
            SkCanvas* canvas = surface->getCanvas();

            // Draw using Skia functions
            SkPaint paint;
            paint.setColor(SK_ColorBLUE);
            canvas->drawRect(SkRect::MakeLTRB(50, 50, 150, 150), paint);

            // Clean up
            EndPaint(hwnd, &ps);
            return 0;
        }
        case WM_CLOSE:
            DestroyWindow(hwnd);
            return 0;
        case WM_DESTROY:
            PostQuitMessage(0);
            return 0;
        default:
            return DefWindowProc(hwnd, msg, wParam, lParam);
    }
}
 

 
//unsigned

int main()
{

    WNDCLASS wc = {0};
    wc.lpfnWndProc = WndProc;
    wc.hInstance = GetModuleHandle(NULL);
    wc.hbrBackground = (HBRUSH)(COLOR_BACKGROUND);
    wc.lpszClassName = "SkiaWindowClass";
    RegisterClass(&wc);

    // Create window
    HWND hwnd = CreateWindow("SkiaWindowClass", "Skia Window", WS_OVERLAPPEDWINDOW,
                             CW_USEDEFAULT, CW_USEDEFAULT, 800, 600,
                             NULL, NULL, wc.hInstance, NULL);
    if (!hwnd) {
        return -1;
    }

    ShowWindow(hwnd, SW_SHOW);

    MSG msg;
    while (GetMessage(&msg, NULL, 0, 0)) {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
    }

    return msg.wParam;

    SKIA();
    float floatmin = FLT_MIN;

    printf("Minimal float value: %.100f \n", FLT_MAX);
    printf("Size of float variable: %zu bytes\n", sizeof(FLT_MIN));

    Channel chan = {};

    AddOutputInput(&chan, 3);
    AddOutputInput(&chan, 3); 
    //AddOutputInput(&chan, 16);
    AddOutputInput(&chan, 2);


    float inputs[5] = { 1,3,4 };
    float inputs2[5] = { 3,0,5 };
   // Inputs* forwardInputs = FloatToInputs(inputs, 5);


    float* inputsNormalizedDeltas = NormalizeDeltas(inputs, 3);
    float* inputsNormalizedDeltas2 = NormalizeDeltas(inputs2, 3);

    Inputs forwardIn = FloatToInputs(inputsNormalizedDeltas, 3);
    Inputs forwardIn2 = FloatToInputs(inputsNormalizedDeltas2, 3);


      
    LogInput(&forwardIn);
    LogInput(&forwardIn2); 


    ////////////////////////////////////
    float trainDesiredOutputs[3] = {  0 , 1 };
    float trainDesiredOutputs2[3] = { 1 , 0 };

    Inputs trainDesiredOutputsForwardIn = FloatToInputs(trainDesiredOutputs, 2); 

    Inputs trainDesiredOutputsForwardIn2 = FloatToInputs(trainDesiredOutputs2, 2);
    ////////////////////////////////////
    /*for (int i = 0; i < 4; i++) {
         MakeFillAllocatedOutputs(&chan, 0);
         Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 0.01);
    }

    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);

    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2);   
    */

    /*
    MakeFillAllocatedOutputs(&chan, 0);
    Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 1);
    /////////////////////////////////////////////////////////////////////////
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);

    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2);
    */
    //LogEroor(&forwardResult4, trainDesiredOutputsForwardIn.allocatedInputs);

     for (int i = 0; i < 5000; i++) {

     MakeFillAllocatedOutputs(&chan, 0);
     Train(&chan, &forwardIn, &trainDesiredOutputsForwardIn, 0.1f);
     //////////////////////////////////////////////
     MakeFillAllocatedOutputs(&chan, 0); 
     Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 0.1f);
     MakeFillAllocatedOutputs(&chan, 0);
    }; 


    //////////////////////////////////////////////
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult3 = ForWards(&chan, &forwardIn);
    LogInput(&forwardResult3);
    LogInput(&trainDesiredOutputsForwardIn);


    //////////////// 
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);
    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2); 
    //////////////////////////////////////////////
     
    hipError_t cudaStatus = hipGetLastError();

    printf("CUDA ERROR: %d\n", cudaStatus != hipSuccess);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaStatus));
        // Handle or report the error appropriately
    }



    return 0;

}