#include <iostream>
#include <stdio.h>
// #include <structs.cu>
#include "channel.cu"

#include <stdio.h>
#include <stdarg.h>

// typedef struct Model
// {
//     Inputs *inputs;
//     Connects *connects;
//     int count;
// } Model;

int main()
{
    float sizes[] = {3, 5, 2};
    Inputs st = {
        sizes,
        3,
    };
    // printf("sizes: %d\n", sizes);

    // Channel chan = {};
    Channel *chan = NewGpuAllocateChannel(1);

    int threadsPerBlock = 1;
    int blocksPerGrid = 1;
    
    AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);
    AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 4);
    AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 4);
    // AddOutputInput(chan, 3);


    
    // AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);
    // AddOutputInput(chan, 3);
    // AddOutputInput(&chan, 3);
    // AddOutputInput(&chan, 4);
    // AddOutputInput(&chan, 4);

    // AddOutputInput(&chan, 4);

    Inputs *res = &st;
    // ForWards(&chan, res);

    // int threadsPerBlock = 1;
    // int blocksPerGrid = 1;
    ForWards<<<blocksPerGrid, threadsPerBlock>>>(chan, res);

    //   int threadsPerBlock = 256; // You can adjust this based on your GPU's capabilities
    // int numBlocks = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    // searchElementKernel<<<numBlocks, threadsPerBlock>>>(hipArray, arraySize, targetElement, cudaElementFound);

    // for (int connectIndex = 0; connectIndex < res->count; connectIndex++)
    // {

    //     // float vv = res->allocatedInputs[connectIndex];
    //     // printf("sizes: %d %f\n", connectIndex, vv);
    //     printf("sizes: %d  \n", connectIndex);
    // }
    // return 0;
}