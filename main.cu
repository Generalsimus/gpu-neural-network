﻿
#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <stdio.h>

int main()
{

    Channel chan = {};

    AddOutputInput(&chan, 3);
    AddOutputInput(&chan, 3);
    AddOutputInput(&chan, 3);
    AddOutputInput(&chan, 3);


    Channel* gpuChan;

    hipMalloc(&gpuChan, sizeof(Channel));
    hipMemcpy(gpuChan, &chan, sizeof(Channel), hipMemcpyHostToDevice);



    printf("{1,2,3,4,5} + {10,20,30,40,50} =  \n");


    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    float sizes[] = { 3, 5, 2 };
    Inputs inputs = {
     sizes,
     3,
   };

    Inputs* forwardOutput = &inputs;
    
    ForWards<<<blocksPerGrid, threadsPerBlock>>>(&chan, forwardOutput);



     hipDeviceSynchronize();

    // Check for errors
     hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
     }
    //Channel* chan = NewGpuAllocateChannel(1);

    //int threadsPerBlock = 1;
    //int blocksPerGrid = 1;
    // 

    //AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);
    //AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);
    //AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);
    //AddOutputInput<<<blocksPerGrid, threadsPerBlock>>>(chan, 3);


    //printf("{1,2,3,4,5} + {10,20,30,40,50} =  \n");

   // float sizes[] = { 3, 5, 2 };
   // Inputs inputs = {
  //      sizes,
  //      3,
  //  };

  //  Inputs* gpuRes;

  //  hipMalloc(&gpuRes, sizeof(Inputs));
  //  hipMemcpy(gpuRes, &inputs, sizeof(Inputs), hipMemcpyHostToDevice);

   // ForWards<<<blocksPerGrid, threadsPerBlock>>> (chan, gpuRes);

  //  Inputs* cpuOut;
    // hipMalloc(&cpuOut, sizeof(Inputs));

 //   hipMemcpy(&cpuOut, gpuRes, sizeof(Inputs), hipMemcpyDeviceToHost);

  //  printf("{1,2,3,4,5} + {10,20,30,40,50} = %d \n", cpuOut->count);
//    for (int outputIndex = 0; outputIndex < cpuOut->count; outputIndex++)
//    {
//        printf("Element %d: %.2f\n", outputIndex, cpuOut->allocatedInputs[outputIndex]);
//    }
     
    return 0;
}