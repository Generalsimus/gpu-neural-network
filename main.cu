#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <stdio.h>
#include <cmath>

 





//uint3 __device_builtin__ __STORAGE__ threadIdx;
//uint3 __device_builtin__ __STORAGE__ blockIdx;
//dim3 __device_builtin__ __STORAGE__ blockDim;
//dim3 __device_builtin__ __STORAGE__ gridDim;
//int __device_builtin__ __STORAGE__ warpSize;
__global__ void ForwardKernel(float* inputs, int inputSize, float* output, int outputSize, float* widths)
{
    unsigned int widthIndex = (blockDim.x * blockIdx.x) + threadIdx.x;

    unsigned int inputIndex = widthIndex / outputSize;
    unsigned int outputIndex = widthIndex - (inputIndex * outputSize);


    output[outputIndex] += inputs[inputIndex] * widths[widthIndex];

   printf("widthIndex: %d \n", widthIndex);
   // printf("inputSize: %d \n", inputSize);
    //printf("inputIndex: %d, outputIndex: %d\n", inputIndex, outputIndex);
}
__global__ void SigmoidKernel(float* output, float* biases)
{
    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;



    output[outputIndex] = 1.0f / (1.0f + expf(-(output[outputIndex] + biases[outputIndex])));

    // printf("outputIndex: %d \n", outputIndex);
    //printf("output: %.2f\n", output[outputIndex]);
}
__global__ void JustForwardKernel(float* inputs, int inputSize, float* outputs, int outputSize, float* widths, float* biases)
{


    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;


    float output = 0;


    for (int inputIndex = 0; inputIndex < inputSize; ++inputIndex) {
        float width = widths[inputIndex + (outputIndex * inputSize)];
        output += width + inputs[inputIndex];
    }



    outputs[outputIndex] = output;
    // printf("outputIndex: %d \n", outputIndex);
    //printf("output: %.2f\n", outputs[outputIndex]);
};

int* CalctThreadsAndBlocks(int num) {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    if (num <= deviceProp.maxThreadsPerBlock) {
        int res[2] = {1,num};
        return res;
    }
    int threads = num % deviceProp.maxThreadsPerBlock;
    int res[2] = { ((num - threads) / deviceProp.maxThreadsPerBlock) + 1,threads};
    return res;
};

__global__ void TestKernel()
{
    unsigned int index = (blockDim.x * blockIdx.x) + threadIdx.x;

    printf("index: %d \n", index);

};

int main()
{
    int miniBatchSize = 111;
    // Set the dimensions of the CUDA grid and blocks
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Assuming device 0
    int maxThreadsXx = prop.maxThreadsDim[0];
    int blockDimX = maxThreadsXx;
    int numBlocksX = (miniBatchSize + blockDimX - 1) / blockDimX;

    printf("blockDimX: %d \n", blockDimX);
    printf("numBlocksX: %d \n", numBlocksX);

    int inputsz = 5;
    int outputsz = 5;


    int* siz = CalctThreadsAndBlocks(inputsz* outputsz);


    int block = siz[0];
    int threads = siz[1];


    TestKernel<<<block, threads>>>();

    printf("block: %d \n", block);
    printf("threads: %d \n", threads);
    return 0;
     const int inputsize = 200000;
     const int outputsize = 30000;


     float* input, *output, *widts, *biases;
     hipMalloc((void**)&input, inputsize * sizeof(float));
     hipMalloc((void**)&output, outputsize * sizeof(float));
     hipMalloc((void**)&widts, inputsize* outputsize * sizeof(float));
     hipMalloc((void**)&biases, outputsize * sizeof(float));

     ////////////////////////////////////////////////////
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);

     // Start recording the execution time
     hipEventRecord(start);
     ////////////////////////////////////////////////////

     //(inputsize * outputsize)

     //ForwardKernel<<<(2147483647 * 50009001), 1 >> >(input, inputsize, output, outputsize, widts);
     SigmoidKernel<<<1, outputsize>>>(output, biases);
     ////////////////////////////////////////////////////
      // Stop recording the execution time
     hipEventRecord(stop);
     hipEventSynchronize(stop);

     // Calculate the elapsed time
     float milliseconds = 0.0f;
     hipEventElapsedTime(&milliseconds, start, stop);

     // Print the kernel execution time
     std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

     // Destroy CUDA events
     hipEventDestroy(start);
     hipEventDestroy(stop);
     //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
     hipEvent_t start2, stop2;
     hipEventCreate(&start2);
     hipEventCreate(&stop2);

     // Start recording the execution time
     hipEventRecord(start2);
     ////////////////////////////////////////////////////

     JustForwardKernel<<<1, outputsize>>>(input, inputsize, output, outputsize, widts, biases);
     ////////////////////////////////////////////////////
      // Stop recording the execution time
     hipEventRecord(stop2);
     hipEventSynchronize(stop2);

     // Calculate the elapsed time
     float milliseconds2 = 0.0f;
     hipEventElapsedTime(&milliseconds2, start2, stop2);

     // Print the kernel execution time
     std::cout << "Kernel execution time22: " << milliseconds2 << " ms" << std::endl;

     // Destroy CUDA events
     hipEventDestroy(start2);
     hipEventDestroy(stop2);

     //////////////////////////////////////////////////// 


      


     int device;
     hipGetDevice(&device);

     hipDeviceProp_t devicePropp;
     hipGetDeviceProperties(&devicePropp, device);

     // Get the maximum number of threads per block
     int maxThreadsPerBlock = devicePropp.maxThreadsPerBlock;

     // Get the maximum dimensions of the grid
     //dim3 maxGridSize = devicePropp.maxGridSize;


     // Get the maximum dimensions of the grid
     int maxBlocksPerMultiprocessor;
     hipDeviceGetAttribute(&maxBlocksPerMultiprocessor, hipDeviceAttributeMaxBlocksPerMultiprocessor, device);

      
    // printf("maxBlocksPerGrid123: %d\n", maxBlocksPerMultiprocessor);

     //myKernel << <maxBlocksPerGrid, maxThreadsPerBlock >> > ();

     int deviceCount;
     hipGetDeviceCount(&deviceCount);

     if (deviceCount == 0) {
         std::cerr << "No CUDA devices found" << std::endl;
         return 1;
     }

     for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
         hipDeviceProp_t deviceProp;
         hipGetDeviceProperties(&deviceProp, deviceId);

         std::cout << "Device ID: " << deviceId << std::endl;
         std::cout << "Device Name: " << deviceProp.name << std::endl;
         std::cout << "Max Blocks per SM: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
         std::cout << "Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
         std::cout << "Max Threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
         std::cout << "Max Grid Size (x, y, z): " << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1]
             << ", " << deviceProp.maxGridSize[2] << std::endl;
         std::cout << std::endl;
    }
    return 0;
}