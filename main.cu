#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "channel.cu"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

 

 
//__global__ void ForwardKernel(float* inputs, int inputSize, float* output, int outputSize, float* widths)
//{
//    unsigned int widthIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//    unsigned int inputIndex = widthIndex / outputSize;
//    unsigned int outputIndex = widthIndex - (inputIndex * outputSize);
//
//
//    output[outputIndex] += inputs[inputIndex] * widths[widthIndex];
//
//   printf("widthIndex: %d \n", widthIndex);
//   // printf("inputSize: %d \n", inputSize);
//    //printf("inputIndex: %d, outputIndex: %d\n", inputIndex, outputIndex);
//}
//__global__ void SigmoidKernel(float* output, float* biases)
//{
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//
//    output[outputIndex] = 1.0f / (1.0f + expf(-(output[outputIndex] + biases[outputIndex])));
//
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", output[outputIndex]);
//}
//__global__ void JustForwardKernel(float* inputs, int inputSize, float* outputs, int outputSize, float* widths, float* biases)
//{
//
//
//    unsigned int outputIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
//
//
//    float output = 0;
//
//
//    for (int inputIndex = 0; inputIndex < inputSize; ++inputIndex) {
//        float width = widths[inputIndex + (outputIndex * inputSize)];
//        output += width + inputs[inputIndex];
//    }
//
//
//
//    outputs[outputIndex] = output;
//    // printf("outputIndex: %d \n", outputIndex);
//    //printf("output: %.2f\n", outputs[outputIndex]);
//};

 
 

 
//unsigned

int main()
{

    float floatmin = FLT_MIN;

    printf("Minimal float value: %.100f \n", FLT_MAX);
    printf("Size of float variable: %zu bytes\n", sizeof(FLT_MIN));

    Channel chan = {};

    AddOutputInput(&chan, 3);
    AddOutputInput(&chan, 16);
    AddOutputInput(&chan, 2);


    float inputs[5] = { 1,3,4 };
    float inputs2[5] = { 3,0,5 };
   // Inputs* forwardInputs = FloatToInputs(inputs, 5);


    float* inputsNormalizedDeltas = NormalizeDeltas(inputs, 3);
    float* inputsNormalizedDeltas2 = NormalizeDeltas(inputs2, 3);

    Inputs forwardIn = FloatToInputs(inputsNormalizedDeltas, 3);
    Inputs forwardIn2 = FloatToInputs(inputsNormalizedDeltas2, 3);


      
    LogInput(&forwardIn);
    LogInput(&forwardIn2); 


    ////////////////////////////////////
    float trainDesiredOutputs[3] = {  0 , 1 };
    float trainDesiredOutputs2[3] = { 1 , 0 };

    Inputs trainDesiredOutputsForwardIn = FloatToInputs(trainDesiredOutputs, 2); 

    Inputs trainDesiredOutputsForwardIn2 = FloatToInputs(trainDesiredOutputs2, 2);
    ////////////////////////////////////
    /*for (int i = 0; i < 4; i++) {
         MakeFillAllocatedOutputs(&chan, 0);
         Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 0.01);
    }

    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);

    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2);   
    */

    /*
    MakeFillAllocatedOutputs(&chan, 0);
    Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 1);
    /////////////////////////////////////////////////////////////////////////
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);

    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2);
    */
    //LogEroor(&forwardResult4, trainDesiredOutputsForwardIn.allocatedInputs);

     for (int i = 0; i < 10000; i++) {

     MakeFillAllocatedOutputs(&chan, 0);
     Train(&chan, &forwardIn, &trainDesiredOutputsForwardIn, 0.1f);
     //////////////////////////////////////////////
     MakeFillAllocatedOutputs(&chan, 0); 
     Train(&chan, &forwardIn2, &trainDesiredOutputsForwardIn2, 0.1f);
     MakeFillAllocatedOutputs(&chan, 0);
    }; 


    //////////////////////////////////////////////
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult3 = ForWards(&chan, &forwardIn);
    LogInput(&forwardResult3);
    LogInput(&trainDesiredOutputsForwardIn);


    //////////////// 
    MakeFillAllocatedOutputs(&chan, 0);
    Inputs forwardResult4 = ForWards(&chan, &forwardIn2);
    LogInput(&forwardResult4);
    LogInput(&trainDesiredOutputsForwardIn2); 
    //////////////////////////////////////////////
     
    hipError_t cudaStatus = hipGetLastError();

    printf("CUDA ERROR: %d\n", cudaStatus != hipSuccess);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaStatus));
        // Handle or report the error appropriately
    }
   
    return 0;


}