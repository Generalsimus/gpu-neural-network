#include "hip/hip_runtime.h"
#include "utils.cu"



typedef struct Inputs
{
    float* allocatedInputs;
    int count;
} Inputs;



__device__ Inputs* NewGpuAllocateInputs(int size)
{
    Inputs* inputs;
    hipMalloc((void**)&inputs, size * sizeof(Inputs));
    return inputs;
};
 
 
