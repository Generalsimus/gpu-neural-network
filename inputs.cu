#include "hip/hip_runtime.h"
#include <corecrt_malloc.h>

typedef struct Inputs
{
    float* allocatedInputs;
    size_t* size;

    dim3 blocksPerGrid;
    dim3 threadsPerBlock;
} Inputs;



void FillInputsDefaultValue(Inputs* inputs, float defaultValue)
{
    /////////////////////////////////////////////////////////////
    float* defaultValueDevice;

    hipMalloc((void**)&defaultValueDevice, sizeof(float));

    hipMemcpy(defaultValueDevice, &defaultValue, sizeof(float), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////// 
    size_t inputSize;


    hipMemcpy(&inputSize, inputs->size, sizeof(size_t), hipMemcpyDeviceToHost);
    /////////////////////////////////////////////////////////////

    AllocateArrayInGpuWithDefaultValue<<<inputs->blocksPerGrid, inputs->threadsPerBlock>>>(inputs->allocatedInputs, inputs->size, defaultValueDevice);
    /////////////////////////////////////////////////////////////
};

Inputs NewInputs(size_t size)
{
    ////////////////////////////////
    size_t* sizeDevice;
    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    ////////////////////////////////  
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));
    ////////////////////////////////
    size_t thredBalance = FindBalanceThread(size);
     

    dim3 blocksPerGrid(size / thredBalance);
    dim3 threadsPerBlock(thredBalance); 
    ////////////////////////////////
    Inputs input = {
        inputDevice,
        sizeDevice,
        blocksPerGrid,
        threadsPerBlock, 
    };
     
    /////////////////////////////////

    return input;
};

Inputs FloatToInputs(float* inputs,const size_t size)
{  
    ////////////////////////////////
    size_t* sizeDevice;

    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    //////////////////////////////// 
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));

    hipMemcpy(inputDevice, inputs, size * sizeof(float), hipMemcpyHostToDevice);
    ////////////////////////////////
    size_t thredBalance = FindBalanceThread(size);


    dim3 blocksPerGrid(size / thredBalance);
    dim3 threadsPerBlock(thredBalance);
    ////////////////////////////////
    Inputs input = {
        inputDevice,
        sizeDevice,
        blocksPerGrid,
        threadsPerBlock,
    }; 
    return input;
};

void LogInput(Inputs* inputs) {

     LogGpuFloatArray<<<inputs->blocksPerGrid, inputs->threadsPerBlock>>>(inputs->allocatedInputs, inputs->size);
};
