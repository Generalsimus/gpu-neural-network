#include "hip/hip_runtime.h"



typedef struct Inputs
{
    float* allocatedInputs;
    size_t* count;
} Inputs;



void FillInputsDefaultValue(Inputs* inputs, float defaultValue)
{
    /////////////////////////////////////////////////////////////
    float* defaultValueDevice;

    hipMalloc((void**)&defaultValueDevice, sizeof(float));

    hipMemcpy(defaultValueDevice, &defaultValue, sizeof(float), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////// 
    size_t inputSize;


    hipMemcpy(&inputSize, inputs->count, sizeof(size_t), hipMemcpyDeviceToHost);
    /////////////////////////////////////////////////////////////

    printf("inputSize: %d \n", inputSize);
    size_t thredBalance = FindBalanceThread(inputSize);


    dim3 blocksPerGrid(inputSize / thredBalance);
    dim3 threadsPerBlock(thredBalance);

    AllocateArrayInGpuWithDefaultValue<<<blocksPerGrid, threadsPerBlock>>>(inputs->allocatedInputs, inputs->count, defaultValueDevice);
    /////////////////////////////////////////////////////////////
};

Inputs NewInputs(size_t size)
{
    ////////////////////////////////
    size_t* sizeDevice;
    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    ////////////////////////////////  
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));
    ////////////////////////////////
    Inputs input = {
        inputDevice,
        sizeDevice,
    };

    FillInputsDefaultValue(&input, 0.5);
    /////////////////////////////////

    return input;
};

Inputs* FloatToInputs(float* inputs, size_t size)
{
    ////////////////////////////////
    size_t* sizeDevice;

    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    ////////////////////////////////  
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));

    hipMemcpy(inputDevice, inputs, size * sizeof(float), hipMemcpyHostToDevice);
    ////////////////////////////////

    Inputs input = {
        inputDevice,
        sizeDevice,
    };

    return &input;
};
