#include "hip/hip_runtime.h"
#include <corecrt_malloc.h>

typedef struct Inputs
{
    float* allocatedInputs;
    size_t* size;
    dim3 blocksPerGrid;
    dim3 threadsPerBlock;
} Inputs;



void FillInputsDefaultValue(Inputs* inputs, float defaultValue)
{
    /////////////////////////////////////////////////////////////
    float* defaultValueDevice;

    hipMalloc((void**)&defaultValueDevice, sizeof(float));

    hipMemcpy(defaultValueDevice, &defaultValue, sizeof(float), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////// 
    size_t inputSize;


    hipMemcpy(&inputSize, inputs->size, sizeof(size_t), hipMemcpyDeviceToHost);
    /////////////////////////////////////////////////////////////

    size_t thredBalance = FindBalanceThread(inputSize);


    dim3 blocksPerGrid(inputSize / thredBalance);
    dim3 threadsPerBlock(thredBalance);
    printf("blocksPerGrid: %d \n", inputSize / thredBalance);
    printf("threadsPerBlock: %d \n", thredBalance);

    AllocateArrayInGpuWithDefaultValue<<<blocksPerGrid, threadsPerBlock>>>(inputs->allocatedInputs, inputs->size, defaultValueDevice);
    /////////////////////////////////////////////////////////////
};

Inputs NewInputs(size_t size)
{
    ////////////////////////////////
    size_t* sizeDevice;
    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    ////////////////////////////////  
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));
    ////////////////////////////////
   // size_t thredBalance = FindBalanceThread(sizeDevice);
    /*ize_t thredBalance = FindBalanceThread(sizeDevice);

    dim3 blocksPerGrid(sizeDevice / thredBalance);
    dim3 threadsPerBlock(thredBalance); */
    ////////////////////////////////
    Inputs input = {
        inputDevice,
        sizeDevice,
       /* blocksPerGrid,
        threadsPerBlock,*/
    };





    //FillInputsDefaultValue(&input, 0.5);
    /////////////////////////////////

    return input;
};

Inputs FloatToInputs(float* inputs,const size_t size)
{  
    ////////////////////////////////
    size_t* sizeDevice;

    hipMalloc((void**)&sizeDevice, sizeof(size_t));

    hipMemcpy(sizeDevice, &size, sizeof(size_t), hipMemcpyHostToDevice);
    //////////////////////////////// 
    float* inputDevice;

    hipMalloc((void**)&inputDevice, size * sizeof(float));

    hipMemcpy(inputDevice, inputs, size * sizeof(float), hipMemcpyHostToDevice);
    ////////////////////////////////
    Inputs input = {
        inputDevice,
        sizeDevice,
    }; 
    return input;
};
