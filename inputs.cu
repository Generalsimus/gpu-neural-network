#include "hip/hip_runtime.h"

typedef struct Inputs
{
    float* allocatedInputs;
    int count;
} Inputs;



__device__ Inputs* NewGpuAllocateInputs(int size)
{
    Inputs* inputs;
    hipMalloc(&inputs, size * sizeof(Inputs));
    return inputs;
};

__device__ Inputs* NewGpuAllocateSingleInputs(int size)
{

    float* inputsValues;

    hipMalloc(&inputsValues, size * sizeof(float));

    Inputs* inputs;
    hipMalloc(&inputs, sizeof(Inputs));

    (*inputs).allocatedInputs = inputsValues;
    (*inputs).count = size;

    return inputs;
};