// cuda.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cudaKernel() {
    printf("Hello from CUDA kernel!\n");
}

extern "C" {
     void runCudaCode() {
        cudaKernel<<<1, 1>>>();
        hipDeviceSynchronize();
    }
}