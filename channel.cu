#include "hip/hip_runtime.h"
#include "utils.cu"
#include "connects.cu"
#include "inputs.cu"
#include <hip/hip_runtime.h>


typedef struct Channel
{
    Inputs* allocatedOutputs;
    Connects* allocatedConnects;
    size_t outputLayerSize;
    size_t layersCount;
} Channel;
 

/* if (chanIndex == 0) {
      printf("START\n ");
      LogInput(forwardInput);
      printf("END\n ");
  }*/
Inputs ForWardAfterIndex(Channel* chan, Inputs* forwardInput, int chanIndex)
{
    if (chanIndex > (chan->layersCount - 2)) {
        printf("Train Error: Channel max size is %d (line %d): %s\n", (chan->layersCount - 1), __LINE__, __FILE__);
    };

    Connects connect = chan->allocatedConnects[chanIndex];
    Inputs outputs = chan->allocatedOutputs[chanIndex];
    /////////////////////////////////////////////////////////////////////////////////////
    

    if (chanIndex == 0) {
        LogInput(forwardInput);
    }

    ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);

     
    ForwardSigmoid<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, connect.biases);
     


    if (chanIndex == (chan->layersCount - 2)) { 
        return outputs;
    }
    else {
        return ForWardAfterIndex(chan, &outputs, (chanIndex + 1));
    }
}

Inputs ForWards(Channel* chan, Inputs* forwardInput)
{
    return ForWardAfterIndex(chan, forwardInput, 0);
};

float* TrainAfterIndex(Channel* chan, Inputs* forwardInput, Inputs* desiredOutputs, float* learnRate, int chanIndex)
{
    if (chanIndex > (chan->layersCount - 2)) {
        printf("Train Error: Channel max size is %d (line %d): %s\n", (chan->layersCount - 1), __LINE__, __FILE__);
    };
    Connects connect = chan->allocatedConnects[chanIndex];
    Inputs outputs = chan->allocatedOutputs[chanIndex];
    /////////////////////////////////////////////////////////////////////////////////////

    ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);

    ForwardSigmoid<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, connect.biases);

    /////////////////////////////////////////////////////////////////////////////////////
    size_t outputsSize = InputSize(&outputs);


    size_t inputsSize = InputSize(forwardInput); 
    /////////////////////////////////////////////////////////////////////////////////////

    //if (chanIndex == 0) {

        //LogInput(forwardInput);
    //}

    if (chanIndex == (chan->layersCount - 2)) {
        float* deltas = AllocateGpuFloatArray(outputsSize); 

        LogError(&outputs, desiredOutputs->allocatedInputs);
        
        TrainError<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, desiredOutputs->allocatedInputs, deltas);
        
        ForwardSigmoidDerivative<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, deltas);

        float* deltasOutputs = AllocateGpuFloatArray(inputsSize);

        TrainUpdateWidths<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths, connect.biases, deltas, deltasOutputs, learnRate);
        

        hipFree(deltas);
        return deltasOutputs;
    }
    else {
        float* deltas = TrainAfterIndex(chan, &outputs, desiredOutputs, learnRate, (chanIndex + 1));
        
        ForwardSigmoidDerivative<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, deltas);

        float* deltasOutputs = AllocateGpuFloatArray(inputsSize);
        //LogGpuFloatArray<<<forwardInput->blocksPerGrid, forwardInput->threadsPerBlock>>>(connect.widths, forwardInput->size);

        TrainUpdateWidths<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths, connect.biases, deltas, deltasOutputs, learnRate);
        
        //LogGpuFloatArray<<<forwardInput->blocksPerGrid, forwardInput->threadsPerBlock>>>(connect.widths, forwardInput->size);
        hipFree(deltas);
        return deltasOutputs;
    }

}
void Train(Channel* chan, Inputs* forwardInput, Inputs* desiredOutputs, float learnRate)
{

    float* learnRateGpu;

    hipMalloc((void**)&learnRateGpu, sizeof(float));

    hipMemcpy(learnRateGpu, &learnRate, sizeof(int), hipMemcpyHostToDevice);


    float* deltas = TrainAfterIndex(chan, forwardInput, desiredOutputs, learnRateGpu, 0);

    hipFree(deltas);
    hipFree(learnRateGpu);
};

void MakeFillAllocatedOutputs(Channel* chan, float defaultValue)
{
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        Inputs outputs = chan->allocatedOutputs[connectIndex];
         
       FillInputsDefaultValue(&outputs, defaultValue);
       
    }
}

void AddOutputInput(Channel* chan, size_t inputSize)
{
    size_t layersCount = chan->layersCount;
   // printf("layersCount: %d \n", layersCount);
     
    if (layersCount > 0) {
        size_t connectsCount = layersCount - 1;


        chan->allocatedConnects = AddElement(chan->allocatedConnects, connectsCount, NewConnection(chan->outputLayerSize, inputSize));

        chan->allocatedOutputs = AddElement(chan->allocatedOutputs, connectsCount, NewInputs(inputSize));

    };


    chan->outputLayerSize = inputSize;
    chan->layersCount = layersCount + 1;
}; 