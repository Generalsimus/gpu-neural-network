#include "utils.cu"
#include "connects.cu"
#include "inputs.cu"
#include <hip/hip_runtime.h>


typedef struct Channel
{
    Inputs* allocatedOutputs;
    Connects* allocatedConnects;
    size_t outputLayerSize;
    size_t layersCount;
} Channel;
 

void ForWards(Channel* chan, Inputs* forwardInput)
{ 
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        printf("connectIndex: %d \n", connectIndex);
        Connects connect = chan->allocatedConnects[connectIndex];
        Inputs outputs = chan->allocatedOutputs[connectIndex];

       ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);
       *forwardInput = outputs;
    }
};
void MakeFillAllocatedOutputs(Channel* chan, float defaultValue)
{
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        Inputs outputs = chan->allocatedOutputs[connectIndex];

        printf("III: %d\n", connectIndex);
        FillInputsDefaultValue(&outputs, defaultValue);
    }
}

void AddOutputInput(Channel* chan, size_t inputSize)
{
    size_t layersCount = chan->layersCount;
    printf("layersCount: %d \n", layersCount);
     
    if (layersCount > 0) {
        size_t index = layersCount - 1;




        chan->allocatedConnects = AddElement(chan->allocatedConnects, index, NewConnection(chan->outputLayerSize, inputSize));

        chan->allocatedOutputs = AddElement(chan->allocatedOutputs, index, NewInputs(inputSize));
        

    };


    chan->outputLayerSize = inputSize;
    chan->layersCount = layersCount + 1;
}; 