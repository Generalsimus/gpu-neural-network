#include "hip/hip_runtime.h"
#include "inputs.cu"
#include "connects.cu"
#include <hip/hip_runtime.h>


typedef struct Channel
{
    Inputs* allocatedOutputs;
    Connects* allocatedConnects;
    int outputLayerSize;
    int layersCount;
} Channel;

__global__ void ForWards(Channel* chan, Inputs* forwardInput)
{ 
};

void AddOutputInput(Channel* chan, int inputSize)
{
    int layersCount = chan->layersCount;
    printf("layersCount: %d \n", layersCount);
     
    if (layersCount > 0){

        int connectsCount = layersCount - 1; 

        ////////////////////////////////////////////////////////////////////////////
        Inputs* allocatedOutputs = (Inputs*)malloc(inputSize * sizeof(Inputs));


        for (int outputIndex = 0; outputIndex < connectsCount; outputIndex++)
        { 
            allocatedOutputs[outputIndex] = chan->allocatedOutputs[outputIndex];
        };

        float* inputsValues = (float*)malloc(inputSize * sizeof(float));

        Inputs newInputs = {
        inputsValues,
        inputSize, 
        };
        allocatedOutputs[connectsCount] = newInputs; 
        chan->allocatedOutputs = allocatedOutputs; 
        ////////////////////////////////////////////////////////////////////////////


        Connects* allocatedConnects = (Connects*)malloc(inputSize * sizeof(Connects));
         
        
       

        for (int connectIndex = 0; connectIndex < connectsCount; connectIndex++)
        {
            allocatedConnects[connectIndex] = chan->allocatedConnects[connectIndex];
        };
        int connectsSize = chan->outputLayerSize * inputSize;
        float* widths = (float*)malloc(connectsSize * sizeof(float));
        float* biases = (float*)malloc(inputSize * sizeof(float));
        Connects newConnect = {
        widths,
        biases,
        };
        allocatedConnects[connectsCount] = newConnect;

        chan->allocatedConnects = allocatedConnects;
         
    }


    chan->outputLayerSize = inputSize;
    chan->layersCount = layersCount + 1; 


    printf("ENDDDDDDDDDDDDDD \n" );

    // #if __CUDA_ARCH__ >= 200
    //     printf("inputSize: %d \n", inputSize);
    //
    // #endif
    //     int layersCount = chan->layersCount;
    //
    //     if (layersCount > 0)
    //     {
    //         int connectsCount = layersCount - 1;
    //
    //         ////////////////////////////////////////////////////////
    //         Inputs* allocatedOutputs = NewGpuAllocateInputs(layersCount);
    //
    //         for (int outputIndex = 0; outputIndex < connectsCount; outputIndex++)
    //         {
    //
    //             allocatedOutputs[outputIndex] = chan->allocatedOutputs[outputIndex];
    //         };
    //
    //         Inputs* newInputsElement = NewGpuAllocateSingleInputs(inputSize);
    //
    //         for (int outputIndex = 0; outputIndex < newInputsElement->count; outputIndex++)
    //         {
    // #if __CUDA_ARCH__ >= 200
    //             printf("wwwwwww: %d \n", outputIndex);
    // #endif
    //         }
    //         //   allocatedOutputs[connectsCount] = *newInputsElement;
    //         //   (*chan).allocatedOutputs = allocatedOutputs;
    //
    //         ////////////////////////////////////////////////////////
    //
    //         //  Connects* allocatedConnects = NewGpuAllocateConnects(layersCount);
    //
    //         //  for (int connectIndex = 0; connectIndex < connectsCount; connectIndex++)
    //         // {
    //         //     allocatedConnects[connectIndex] = chan->allocatedConnects[connectIndex];
    //         //  };
    //
    //         //   Connects* connects = CreateConnection((*chan).outputLayerSize, inputSize);
    //         //    allocatedConnects[connectsCount] = *connects;
    //
    //         //  (*chan).allocatedConnects = allocatedConnects;
    //
    //         ////////////////////////////////////////////////////////
    //         // }
    //     };
    //
    //     (*chan).outputLayerSize = inputSize;
    //     (*chan).layersCount = layersCount + 1;
    //
    //     //  hipDeviceSynchronize();
    //     // for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
    //     //  {
    //     //  }/
}; 