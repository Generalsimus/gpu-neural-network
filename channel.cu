#include "hip/hip_runtime.h"

#include "utils.cu"
#include "connects.cu"

////////////////////////
typedef struct Channel
{
    Inputs *allocatedOutputs;
    Connects *allocatedConnects;
    int outputLayerSize;
    int layersCount;
} Channel;

__global__ void ForWards(Channel *chan, Inputs *forwardInput)
{
    for (int connectIndex = 0; connectIndex < ((*chan).layersCount - 1); connectIndex++)
    {

        Connects connect = (*chan).allocatedConnects[connectIndex];
        Inputs outputs = (*chan).allocatedOutputs[connectIndex];

        Forward(&connect, forwardInput, &outputs);

        ////
        *forwardInput = outputs;
    }
}

__global__ void AddOutputInput(Channel *chan, int inputSize)
{

    // std::cout << "Logger Data: ";
    // std::cout << 12 << " ";
    // for (int i = 0; i < dataSize; ++i) {
    //     std::cout << logData[i] << " ";
    // }
    // std::cout << std::endl;
    printf("lDdsds\n");
    int layersCount = chan->layersCount;
    printf("layersCount: %d \n", layersCount);
    if (layersCount > 0)
    {
        int connectsCount = layersCount - 1;

        ////////////////////////////////////////////////////////
        Inputs *allocatedOutputs = NewGpuAllocateInputs(layersCount);
        // (Inputs *)malloc(layersCount * sizeof(Inputs));

        for (int outputIndex = 0; outputIndex < connectsCount; outputIndex++)
        {
            printf("WW: %d \n", outputIndex);
            allocatedOutputs[outputIndex] = chan->allocatedOutputs[outputIndex];
        };

        Inputs *newInputsElement = NewGpuAllocateSingleInputs(inputSize);
        allocatedOutputs[connectsCount] = *newInputsElement;
        (*chan).allocatedOutputs = allocatedOutputs;

        printf("7885: %d \n", newInputsElement->count);

        ////////////////////////////////////////////////////////

        Connects *allocatedConnects = NewGpuAllocateConnects(layersCount);

        for (int connectIndex = 0; connectIndex < connectsCount; connectIndex++)
        {
            allocatedConnects[connectIndex] = chan->allocatedConnects[connectIndex];
        };

        Connects *connects = CreateConnection((*chan).outputLayerSize, inputSize);
        allocatedConnects[connectsCount] = *connects;

        (*chan).allocatedConnects = allocatedConnects;

        ////////////////////////////////////////////////////////
    }

    (*chan).outputLayerSize = inputSize;
    (*chan).layersCount = layersCount + 1;
}

Channel *NewGpuAllocateChannel(int size)
{
    Channel *devicePtr;
    hipMalloc(&devicePtr, size * sizeof(float));
    return devicePtr;
}