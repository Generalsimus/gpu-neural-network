#include "hip/hip_runtime.h"
#include "inputs.cu"
#include "connects.cu"
#include "utils.cu"
#include <hip/hip_runtime.h>

typedef struct Channel
{
    Inputs* allocatedOutputs;
    Connects* allocatedConnects;
    int outputLayerSize;
    int layersCount;
} Channel;

__global__ void ForWards(Channel* chan, Inputs* forwardInput)
{
    for (int connectIndex = 0; connectIndex < ((*chan).layersCount - 1); connectIndex++)
    {

        Connects connect = (*chan).allocatedConnects[connectIndex];
        Inputs outputs = (*chan).allocatedOutputs[connectIndex];

        Forward(&connect, forwardInput, &outputs);

        ////
        *forwardInput = outputs;
    }
}


__global__ void AddOutputInput(Channel* chan, int inputSize)
{
     
    int layersCount = chan->layersCount;

    if (layersCount > 0)
    {
        int connectsCount = layersCount - 1;

        ////////////////////////////////////////////////////////
        Inputs* allocatedOutputs = NewGpuAllocateInputs(layersCount);

        for (int outputIndex = 0; outputIndex < connectsCount; outputIndex++)
        {
           
            allocatedOutputs[outputIndex] = chan->allocatedOutputs[outputIndex];
        };

        Inputs* newInputsElement = NewGpuAllocateSingleInputs(inputSize);
        allocatedOutputs[connectsCount] = *newInputsElement;
        (*chan).allocatedOutputs = allocatedOutputs;
         
        ////////////////////////////////////////////////////////

        Connects* allocatedConnects = NewGpuAllocateConnects(layersCount);

        for (int connectIndex = 0; connectIndex < connectsCount; connectIndex++)
        {
            allocatedConnects[connectIndex] = chan->allocatedConnects[connectIndex];
        };

        Connects* connects = CreateConnection((*chan).outputLayerSize, inputSize);
        allocatedConnects[connectsCount] = *connects;

        (*chan).allocatedConnects = allocatedConnects;

        ////////////////////////////////////////////////////////
    }

    (*chan).outputLayerSize = inputSize;
    (*chan).layersCount = layersCount + 1;
}

Channel* NewGpuAllocateChannel(int size)
 {
   Channel *channel;

    hipMalloc((void**)&channel, size * sizeof(Channel));

    return channel;
 }