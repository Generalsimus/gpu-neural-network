#include "hip/hip_runtime.h"
#include "utils.cu"
#include "connects.cu"
#include "inputs.cu"
#include <hip/hip_runtime.h>


typedef struct Channel
{
    Inputs* allocatedOutputs;
    Connects* allocatedConnects;
    size_t outputLayerSize;
    size_t layersCount;
} Channel;
 

void ForWards(Channel* chan, Inputs* forwardInput)
{
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        printf("connectIndex: %d \n", connectIndex);
        Connects connect = chan->allocatedConnects[connectIndex];
        Inputs outputs = chan->allocatedOutputs[connectIndex];

        ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);

        ForwardSigmoid<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, connect.biases);

        *forwardInput = outputs;
    }
};

float* TrainAfterIndex(Channel* chan, Inputs* forwardInput, Inputs* desiredOutputs, float learnRate,int chanIndex)
{
    if (chanIndex > (chan->layersCount - 1)) {
        printf("Train Error: Channel max size is %d (line %d): %s\n", (chan->layersCount - 1), __LINE__, __FILE__);
    };
    Connects connect = chan->allocatedConnects[chanIndex];
    Inputs outputs = chan->allocatedOutputs[chanIndex];
    /////////////////////////////////////////////////////////////////////////////////////

    ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);

    ForwardSigmoid<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, connect.biases);

    /////////////////////////////////////////////////////////////////////////////////////
    size_t outputsSize;

    hipMemcpy(&outputsSize, outputs.size, sizeof(size_t), hipMemcpyDeviceToHost);

    size_t inputsSize;

    hipMemcpy(&inputsSize, forwardInput->size, sizeof(size_t), hipMemcpyDeviceToHost);
    /////////////////////////////////////////////////////////////////////////////////////
   
    if (chanIndex == (chan->layersCount - 2)) {
        float* deltas = AllocateGpuFloatArray(outputsSize);
        TrainError<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, desiredOutputs->allocatedInputs, deltas);

        float* deltasOutputs = AllocateGpuFloatArray(inputsSize);


        TrainUpdateWidths<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths, connect.biases, deltas, deltasOutputs);

        return deltasOutputs;
    }
    else { 
        float* deltas = TrainAfterIndex(chan, &outputs, desiredOutputs, learnRate, (chanIndex + 1));

        float* deltasOutputs = AllocateGpuFloatArray(inputsSize);

        TrainUpdateWidths<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths, connect.biases, deltas, deltasOutputs);
        return deltasOutputs;
    }

}
void Train(Channel* chan, Inputs* forwardInput, Inputs* desiredOutputs, float learnRate)
{ 
    float* deltas;
    //inputs[]float64, desiredOutputs[]float64, learnRate float64
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        printf("connectIndex: %d \n", connectIndex);
        Connects connect = chan->allocatedConnects[connectIndex];
        Inputs outputs = chan->allocatedOutputs[connectIndex];

       ForwardSum<<<connect.blocksPerGrid, connect.threadsPerBlock>>>(forwardInput->allocatedInputs, forwardInput->size, outputs.allocatedInputs, outputs.size, connect.widths);

       ForwardSigmoid<<<outputs.blocksPerGrid, outputs.threadsPerBlock>>>(outputs.allocatedInputs, connect.biases);
        
       *forwardInput = outputs;
    }

    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
    }

    //if (connectIndex == (chan->layersCount - 2)) {
    //    size_t outputsSize;

    //    hipMemcpy(&outputsSize, outputs.size, sizeof(size_t), hipMemcpyDeviceToHost);

    //    float* newDelta;
    //    hipMalloc((void**)&newDelta, outputsSize * sizeof(float));

    //    //float* outputs, float* desiredOutputs, float* errorAs
    //    TrainError << <outputs.blocksPerGrid, outputs.threadsPerBlock >> > (outputs.allocatedInputs, desiredOutputs.allocatedInputs, newDelta);

    //}
    //else {

    //}
};

void MakeFillAllocatedOutputs(Channel* chan, float defaultValue)
{
    for (int connectIndex = 0; connectIndex < (chan->layersCount - 1); connectIndex++)
    {
        Inputs outputs = chan->allocatedOutputs[connectIndex];

        printf("III: %d\n", connectIndex);
        FillInputsDefaultValue(&outputs, defaultValue);
    }
}

void AddOutputInput(Channel* chan, size_t inputSize)
{
    size_t layersCount = chan->layersCount;
    printf("layersCount: %d \n", layersCount);
     
    if (layersCount > 0) {
        size_t index = layersCount - 1;




        chan->allocatedConnects = AddElement(chan->allocatedConnects, index, NewConnection(chan->outputLayerSize, inputSize));

        chan->allocatedOutputs = AddElement(chan->allocatedOutputs, index, NewInputs(inputSize));
        

    };


    chan->outputLayerSize = inputSize;
    chan->layersCount = layersCount + 1;
}; 