#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

 float* AllocateGpuFloatArray(int size)
{
  //  float* input = (float*)malloc(size * sizeof(float));

    float* d_input;

    hipMalloc((void**)&d_input, size * sizeof(float));

   // cudaMemcpy(d_input, input, size * sizeof(float), cudaMemcpyHostToDevice);

  //  free(input);

    return d_input;
}

#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
 __device__ void cdpAssert(hipError_t code, const char* file, int line, bool abort = true)
 {
     if (code != hipSuccess)
     {
         printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
         if (abort) assert(0);
     }
 }