#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>

#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t cudaStatus = call;                                           \
        if (cudaStatus != hipSuccess) {                                         \
            fprintf(stderr, "CUDA Error: %s (line %d): %s\n", hipGetErrorString(cudaStatus), __LINE__, __FILE__); \
            exit(1);                                                             \
        }                                                                        \
    } while(0);



size_t FindBalanceThread(size_t num)
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);


    for (size_t testNum = deviceProp.maxThreadsPerBlock; testNum != 0; --testNum) {
        if ((num % testNum) == 0) {
            return testNum;
        }
    }

    return 1;
};


template<typename F>
__global__ void AllocateArrayInGpuWithDefaultValue(F *inputs, F *defaultNum)
{
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    inputs[inputIndex] = *defaultNum;
};



template<typename T>
void CudaMemoryFIll(T*allocatedArray, size_t size, T defaultValue)
{
    T* defaultValueDevice;

    hipMalloc((void**)&defaultValueDevice, sizeof(T));

    hipMemcpy(defaultValueDevice, &defaultValue, sizeof(T), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////// 
    size_t thredBalance = FindBalanceThread(size);


    dim3 blocksPerGrid(size / thredBalance);
    dim3 threadsPerBlock(thredBalance);
    /////////////////////////////////////////////////////////////
     
    AllocateArrayInGpuWithDefaultValue<<<blocksPerGrid, threadsPerBlock>>>(allocatedArray, defaultValueDevice);
}


float* AllocateGpuFloatArray(size_t size)
{
    //  float* input = (float*)malloc(size * sizeof(float));

    float* values;
    //printf("EEEE: %d \n",size);

    CUDA_CHECK(hipMalloc((void**)&values, size * sizeof(float)));
    // cudaMemcpy(d_input, input, size * sizeof(float), cudaMemcpyHostToDevice);
   
    CudaMemoryFIll(values, size, 0.00f);

    return values;
};

 


template<typename T>
T* AddElement(T* array, size_t size, T element)
{
    // Create a new array with increased size
    size_t newSize = size + 1;
    T* newArray = (T*)malloc(newSize * sizeof(T));

    // Copy existing elements to the new array
    for (size_t i = 0; i < size; i++)
    {
        newArray[i] = array[i];
    }

    // Append the new element
    newArray[size] = element;

    // Free the memory of the old array
    free(array);

    return newArray;
}
//#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
//template <typename T>
//void check(T err, const char* const func, const char* const file,
//    const int line)
//{
//    if (err != cudaSuccess)
//    {
//        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
//            << std::endl;
//        std::cerr << cudaGetErrorString(err) << " " << func << std::endl;
//        // We don't exit when we encounter CUDA errors in this example.
//        // std::exit(EXIT_FAILURE);
//    }
//}
//template <typename T, typename PropertyType>
//__device__ void setStructProperty(T* myStruct, PropertyType T::* property, PropertyType newValue) {
//    myStruct->*property = newValue;
//}



__global__ void LogGpuFloatArray(float* inputs, size_t* size)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;


    if (index == 0) {
        printf("\n[ %d: %.10f", index, inputs[index]);
    }
    else {
        printf(", %d: %.10f", index, inputs[index]);
    } 

    if (index == (*size - 1)) {

        printf(" ]\n");
    }
}



float* NormalizeDeltas(float *inputs, size_t size)
{ 
    float maxValue = 0.0f;

    for (size_t i = 1; i < size; i++) {
        if (inputs[i] > maxValue) {
            maxValue = inputs[i];
        }
    }

    float* normalizedDeltas = (float*)malloc(size * sizeof(float));

    for (size_t i = 0; i < size; i++) {
        normalizedDeltas[i] = (inputs[i] / maxValue);
    }
     
    return normalizedDeltas;
}

__global__ void GpuSumFloatArray(float *sumAt, float *inputs)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    atomicAdd(sumAt, inputs[index]);
}

float* SumFloatArray(float* inputs, size_t* size)
{
    size_t inputSize;

    CUDA_CHECK(hipMemcpy(&inputSize, size, sizeof(size_t), hipMemcpyDeviceToHost));


    size_t thredBalance = FindBalanceThread(inputSize);


    dim3 blocksPerGrid(inputSize / thredBalance);
    dim3 threadsPerBlock(thredBalance);

    float *sumAt;
    //printf("EEEE: %d \n",size);

    CUDA_CHECK(hipMalloc((void**)&sumAt, sizeof(float)));

    float sumAtHost = 0;

    hipMemcpy(sumAt, &sumAtHost, sizeof(float), hipMemcpyHostToDevice);

    GpuSumFloatArray<<<blocksPerGrid, threadsPerBlock>>>(sumAt, inputs);

    return sumAt;
};

__global__ void GpuDisadvantageSameSizeFloatArrayElements(float* outputs, float* desiredOutputs, float* errorAs) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

  //  printf("outputIndexxxx: %d \n", outputIndex);
    float output = outputs[outputIndex];
    //float outputTanh = tanh(output);
    //sigmoidX* (1 - sigmoidX);
    errorAs[outputIndex] = (desiredOutputs[outputIndex] - output);

   //  printf("EEEE: %.10f-%.10f=%.10f \n", desiredOutputs[outputIndex], output, errorAs[outputIndex]);
   // printf("errorAs[outputIndex]: %.5f : %.5f : %.5f \n", desiredOutputs[outputIndex], output, (desiredOutputs[outputIndex] - output) * (output * (1 - output)));
};

float* DisadvantageSameSizeFloatArrayElements(float* inputs, float* disadvantageInputs, size_t* size)
{

    size_t inputSize;

    hipMemcpy(&inputSize, size, sizeof(size_t), hipMemcpyDeviceToHost);


    size_t thredBalance = FindBalanceThread(inputSize);


    dim3 blocksPerGrid(inputSize / thredBalance);
    dim3 threadsPerBlock(thredBalance);


    float* disadvantagedInputs = AllocateGpuFloatArray(inputSize);

    GpuDisadvantageSameSizeFloatArrayElements<<<blocksPerGrid, threadsPerBlock>>>(inputs, disadvantageInputs, disadvantagedInputs);

    return disadvantagedInputs;
}


__global__ void GpuSquareArrayNumbers(float* inputs, float* outputs) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    float input = inputs[outputIndex];


    outputs[outputIndex] = input * input; 
};

float* SquareArrayNumbers(float* inputs, size_t* size)
{

    size_t inputSize;

    hipMemcpy(&inputSize, size, sizeof(size_t), hipMemcpyDeviceToHost);


    size_t thredBalance = FindBalanceThread(inputSize);


    dim3 blocksPerGrid(inputSize / thredBalance);
    dim3 threadsPerBlock(thredBalance);
     

    float* outputs = AllocateGpuFloatArray(inputSize);
    GpuSquareArrayNumbers<<<blocksPerGrid, threadsPerBlock>>>(inputs, outputs);

    return outputs;
}