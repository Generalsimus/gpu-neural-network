#include "hip/hip_runtime.h"

 float* AllocateGpuFloatArray(int size)
{
  //  float* input = (float*)malloc(size * sizeof(float));

    float* d_input;

    hipMalloc((void**)&d_input, size * sizeof(float));

   // cudaMemcpy(d_input, input, size * sizeof(float), cudaMemcpyHostToDevice);

  //  free(input);

    return d_input;
}
