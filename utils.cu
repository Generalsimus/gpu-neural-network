#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>


float* AllocateGpuFloatArray(size_t size)
{
    //  float* input = (float*)malloc(size * sizeof(float));

    float* d_input;

    hipMalloc((void**)&d_input, size * sizeof(float));

    // cudaMemcpy(d_input, input, size * sizeof(float), cudaMemcpyHostToDevice);

   //  free(input);

    return d_input;
};


template<typename T>
T* AddElement(T* array, size_t size, T element)
{
    // Create a new array with increased size
    size_t newSize = size + 1;
    T* newArray = (T*)malloc(newSize * sizeof(T));

    // Copy existing elements to the new array
    for (size_t i = 0; i < size; i++)
    {
        newArray[i] = array[i];
    }

    // Append the new element
    newArray[size] = element;

    // Free the memory of the old array
    free(array);

    return newArray;
}
//#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
//template <typename T>
//void check(T err, const char* const func, const char* const file,
//    const int line)
//{
//    if (err != cudaSuccess)
//    {
//        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
//            << std::endl;
//        std::cerr << cudaGetErrorString(err) << " " << func << std::endl;
//        // We don't exit when we encounter CUDA errors in this example.
//        // std::exit(EXIT_FAILURE);
//    }
//}
//template <typename T, typename PropertyType>
//__device__ void setStructProperty(T* myStruct, PropertyType T::* property, PropertyType newValue) {
//    myStruct->*property = newValue;
//}

template<typename F>
__global__ void AllocateArrayInGpuWithDefaultValue(F* inputs, size_t* size, F* defaultNum)
{
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    inputs[inputIndex] = *defaultNum;

    // printf("DDD Float value: %d\n", inputIndex);
     ///printf("Fill Float value: %f\n", inputs[inputIndex]);
     // printf("Fill Float value: %f\n", inputs[inputIndex]);
};
 


size_t FindBalanceThread(size_t num)
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);


    for (size_t testNum = deviceProp.maxThreadsPerBlock; testNum != 0; --testNum) {
        if ((num % testNum) == 0) {
            return testNum;
        }
    }

    return 1;
};

#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t cudaStatus = call;                                           \
        if (cudaStatus != hipSuccess) {                                         \
            fprintf(stderr, "CUDA Error: %s (line %d): %s\n", hipGetErrorString(cudaStatus), __LINE__, __FILE__); \
            exit(1);                                                             \
        }                                                                        \
    } while(0)

__global__ void LogGpuFloatArray(float* inputs, size_t* size)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (index == 0) {
        printf("[ %.10f", inputs[index]);
    }
    else {
        printf(", %.10f", inputs[index]);
    }

    if (index == (*size - 1)) {

        printf(" ]\n");
    }
}

