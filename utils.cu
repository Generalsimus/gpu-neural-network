#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>



float* AllocateGpuFloatArray(int size)
{
    //  float* input = (float*)malloc(size * sizeof(float));

    float* d_input;

    hipMalloc((void**)&d_input, size * sizeof(float));

    // cudaMemcpy(d_input, input, size * sizeof(float), cudaMemcpyHostToDevice);

   //  free(input);

    return d_input;
};
//#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
//template <typename T>
//void check(T err, const char* const func, const char* const file,
//    const int line)
//{
//    if (err != cudaSuccess)
//    {
//        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
//            << std::endl;
//        std::cerr << cudaGetErrorString(err) << " " << func << std::endl;
//        // We don't exit when we encounter CUDA errors in this example.
//        // std::exit(EXIT_FAILURE);
//    }
//}
//template <typename T, typename PropertyType>
//__device__ void setStructProperty(T* myStruct, PropertyType T::* property, PropertyType newValue) {
//    myStruct->*property = newValue;
//}

