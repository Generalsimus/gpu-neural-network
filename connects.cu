#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <corecrt_math.h>

typedef struct Connects
{
    float* widths;
    float* biases;

    dim3 blocksPerGrid;
    dim3 threadsPerBlock;

} Connects;



__global__ void ForwardTanh(float* outputs, float* biases) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    float value = outputs[outputIndex] + biases[outputIndex];
    
    outputs[outputIndex] = (expf(value) - expf(-value)) / (expf(value) + expf(-value));
};




__global__ void ForwardSum(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths) {
    size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t widthIndex = inputIndex * *outputsSize + outputIndex;



    outputs[outputIndex] += inputs[inputIndex] * widths[widthIndex];
};

__global__ void ForwardSigmoid(float* outputs, float* biases) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    outputs[outputIndex] = 1.0f / (1.0f + expf(-(outputs[outputIndex] + biases[outputIndex])));
};


__global__ void ForwardSigmoidDerivative(float* inputs, float* deltas) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    float input = inputs[outputIndex];

    deltas[outputIndex] = deltas[outputIndex] * (input * (1 - input));
};



__global__ void TrainError(float* outputs, float* desiredOutputs, float* errorAs) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

  //  printf("outputIndexxxx: %d \n", outputIndex);
    float output = outputs[outputIndex];

    errorAs[outputIndex] = (desiredOutputs[outputIndex] - output) * (output * (1 - output));

   // printf("errorAs[outputIndex]: %.5f : %.5f : %.5f \n", desiredOutputs[outputIndex], output, (desiredOutputs[outputIndex] - output) * (output * (1 - output)));
};

__global__ void TrainUpdateWidths(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths, float* biases, float* deltas, float* deltasOutputs, float* learnRate) {
    size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t widthIndex = inputIndex * *outputsSize + outputIndex;


    //printf("widths1: %.5f \n", widths[widthIndex]);

    widths[widthIndex] -= (deltas[outputIndex] * *learnRate * inputs[inputIndex]);


    //printf("widths2: %.5f \n", widths[widthIndex]);

    deltasOutputs[inputIndex] += (deltas[outputIndex] * widths[widthIndex]);
};



Connects NewConnection(size_t inputSize, size_t outputSize)
{
    /////////////////////////////////////////////
    size_t inputThredBalance = FindBalanceThread(inputSize);
    size_t outputThredBalance = FindBalanceThread(outputSize);


    dim3 blocksPerGrid(inputSize / inputThredBalance, outputSize / outputThredBalance);
    dim3 threadsPerBlock(inputThredBalance, outputThredBalance);
    /////////////////////////////////////////////

    float* widths;
    hipMalloc((void**)&widths, inputSize * outputSize * sizeof(float));

    //////////////////////////////////////////////

    float* biases;
    hipMalloc((void**)&biases, outputSize * sizeof(float));

    //////////////////////////////////////////////

   
    Connects connects = {
        widths,
        biases,
        blocksPerGrid,
        threadsPerBlock,
    };
     
    return connects;
};


