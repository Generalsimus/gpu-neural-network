#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <corecrt_math.h>

typedef struct Connects
{
    float* widths;
    float* biases;

    dim3 blocksPerGrid;
    dim3 threadsPerBlock;

} Connects;



__global__ void ForwardSigmoid(float* outputs, float* biases) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    float value = outputs[outputIndex] + biases[outputIndex];
    
    //1 / (1 + Math.exp(-x));

    outputs[outputIndex] = 1 / (1 + expf(-value));

   // printf("SIGMO outputs[%d]: %.10f :::: %.10f \n", outputIndex, value, outputs[outputIndex]);

};




__global__ void ForwardSum(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths) {
    size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t widthIndex = inputIndex * *outputsSize + outputIndex;


    float bef = outputs[outputIndex];

    //outputs[outputIndex] = outputs[outputIndex] + inputs[inputIndex] * widths[widthIndex];
    atomicAdd(&outputs[outputIndex], inputs[inputIndex] * widths[widthIndex]);

    //printf("outputs[%d]: %.10f :::: %.10f :::: %.10f :::: %.20f \n", outputIndex, outputs[outputIndex], inputs[inputIndex], widths[widthIndex], inputs[inputIndex] * widths[widthIndex]);
   
    //printf("outputs[%d]: %.10f :::: %.10f :::: %.10f \n", outputIndex, outputs[outputIndex], (inputs[inputIndex] * widths[widthIndex]), bef);
};

 


__global__ void ForwardSigmoidDerivative(float* inputs, float* deltas) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    float input = inputs[outputIndex];


    //sigmoidX* (1 - sigmoidX);


   //float inputTanh =  (expf(input) - expf(-input)) / (expf(input) + expf(-input))
    //printf("ER: %.5f ::: %.5f \n", deltas[outputIndex], deltas[outputIndex] * (input * (1 - input)));

    deltas[outputIndex] = deltas[outputIndex] * (input * (1 - input));

};



__global__ void TrainError(float* outputs, float* desiredOutputs, float* deltas) {
    size_t outputIndex = blockIdx.x * blockDim.x + threadIdx.x;

  //  printf("outputIndexxxx: %d \n", outputIndex);
    float output = outputs[outputIndex];
    //float outputTanh = tanh(output);
    //sigmoidX* (1 - sigmoidX);

    deltas[outputIndex] = (  (desiredOutputs[outputIndex] - output));

     //printf("ER: %.5f ::: %.5f \n", deltas[outputIndex], (desiredOutputs[outputIndex] - output));
   // printf("errorAs[outputIndex]: %.5f : %.5f : %.5f \n", desiredOutputs[outputIndex], output, (desiredOutputs[outputIndex] - output) * (output * (1 - output)));
};

__global__ void TrainUpdateWidths(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths, float* biases, float* deltas, float* deltasOutputs, float* learnRate) {
    size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t widthIndex = inputIndex * *outputsSize + outputIndex;


    //printf("widths1: %.5f \n", widths[widthIndex]);

    deltasOutputs[inputIndex] += (deltas[outputIndex] * widths[widthIndex]);


    //widths[widthIndex] += *learnRate * (deltas[outputIndex] * inputs[inputIndex]);
    // 
   //printf("DDDD: %d %.5f \n", widthIndex, (*learnRate * (deltas[outputIndex] * inputs[inputIndex])));

    atomicAdd(&widths[widthIndex], (*learnRate * (deltas[outputIndex] * inputs[inputIndex])));
    //atomicAdd(&widths[widthIndex], (*learnRate * (deltas[outputIndex] * outputs[outputIndex])));


    //atomicAdd(&biases[outputIndex], -(*learnRate * deltas[outputIndex]));
    atomicAdd(&biases[outputIndex], (*learnRate * (deltas[outputIndex] * 1)));
    //atomicAdd(&widths[widthIndex], -(*learnRate * (deltas[outputIndex] * inputs[inputIndex])));

    

    //atomicAdd(&deltasOutputs[inputIndex], (deltas[outputIndex] * inputs[inputIndex]));

    //printf("DDDD: %d :::: %.5f :::: %.5f :::: %.5f :::: %.5f \n", inputIndex, deltas[outputIndex], widths[widthIndex], deltasOutputs[inputIndex], (deltas[outputIndex] * widths[widthIndex]));

    //printf("DDDD: %d %.5f \n", inputIndex, deltasOutputs[inputIndex]);


    //printf("widthIndex: %d \n", widthIndex);
    //printf("widths2: %.5f \n", widths[widthIndex]);

};



Connects NewConnection(size_t inputSize, size_t outputSize)
{
    /////////////////////////////////////////////
    size_t inputThredBalance = FindBalanceThread(inputSize);
    size_t outputThredBalance = FindBalanceThread(outputSize);


    dim3 blocksPerGrid(inputSize / inputThredBalance, outputSize / outputThredBalance);
    dim3 threadsPerBlock(inputThredBalance, outputThredBalance);
    /////////////////////////////////////////////

    float* widths;
    hipMalloc((void**)&widths, inputSize * outputSize * sizeof(float));


    CudaMemoryFIll(widths, inputSize * outputSize, 0.50f);
    //////////////////////////////////////////////

    float* biases;
    hipMalloc((void**)&biases, outputSize * sizeof(float));

    CudaMemoryFIll(biases, outputSize, 1.00f);
    //////////////////////////////////////////////

   
    Connects connects = {
        widths,
        biases,
        blocksPerGrid,
        threadsPerBlock,
    };
     
    return connects;
};


