#include "hip/hip_runtime.h"

typedef struct Connects
{
    float* widths;
    float* biases;
} Connects;

__device__ void Forward(Connects* LayerConnect, Inputs* input, Inputs* output)
{

    int outputSize = output->count;
    int inputSize = input->count;

    float* biases = LayerConnect->biases;
    float* widths = LayerConnect->widths;


    for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
    {

        float outputElement = 0;

        for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
        {
            float inputElement = input->allocatedInputs[inputIndex];

            int widthIndex = ((inputIndex * outputIndex) + outputIndex);

            outputElement += inputElement * widths[widthIndex];
        };
        (*output).allocatedInputs[outputIndex] = outputElement + biases[outputIndex];
    };
}
// ((inputIndex* OutputIndex) + OutputIndex)


__device__ Connects* CreateConnection(int inputSize, int outputSize)
{
    float* widths;

    hipMalloc((void**)&widths, inputSize * outputSize * sizeof(float));

    float* biases;

    hipMalloc((void**)&biases, outputSize * sizeof(float));

    Connects* connects;
    hipMalloc((void**)&connects, sizeof(Connects));

    (*connects).widths = widths;
    (*connects).biases = biases;

    return connects;
};


__device__ Connects* NewGpuAllocateConnects(int size)
{
    Connects* connect;
    hipMalloc((void**)&connect, size * sizeof(Connects));
    return connect;
}