#include "hip/hip_runtime.h"
#include "utils.cu"
#include "hip/hip_runtime.h"
#include ""


typedef struct Connects
{
    float* widths;
    float* biases;
    dim3 blocksPerGrid;
    dim3 threadsPerBlock;
} Connects;


__global__ void ForwardSum(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths) {
    int outputIndex = blockIdx.y * blockDim.y + threadIdx.y; 
    int inputIndex = blockIdx.x * blockDim.x + threadIdx.x;

    printf("outputIndex: %d ,inputIndex: %d \n", outputIndex, inputIndex); 

    printf("Float value: %f\n", inputs[inputIndex]);

}
//__device__ void Forward(Connects* LayerConnect, Inputs* input, Inputs* output)
//{
//
//    int outputSize = output->count;
//    int inputSize = input->count;
//
//    float* biases = LayerConnect->biases;
//    float* widths = LayerConnect->widths;
//
//
//    for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
//    {
//
//        float outputElement = 0;
//
//        for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
//        {
//            float inputElement = input->allocatedInputs[inputIndex];
//
//            int widthIndex = ((inputIndex * outputIndex) + outputIndex);
//
//            outputElement += inputElement * widths[widthIndex];
//        };
//        (*output).allocatedInputs[outputIndex] = outputElement + biases[outputIndex];
//    };
//}
// ((inputIndex* OutputIndex) + OutputIndex)

Connects NewConnection(size_t inputSize, size_t outputSize)
{
    /////////////////////////////////////////////
    size_t inputThredBalance = FindBalanceThread(inputSize);
    size_t outputThredBalance = FindBalanceThread(outputSize);


    dim3 blocksPerGrid(inputSize / inputThredBalance, outputSize / outputThredBalance);
    dim3 threadsPerBlock(inputThredBalance, outputThredBalance);
    /////////////////////////////////////////////

    float* widths;
    hipMalloc((void**)&widths, inputSize * outputSize * sizeof(float));

    //////////////////////////////////////////////

    float* biases;
    hipMalloc((void**)&biases, outputSize * sizeof(float));

    //////////////////////////////////////////////

   
    Connects connects = {
        widths,
        biases,
        blocksPerGrid,
        threadsPerBlock,
    };
     
    return connects;
};


