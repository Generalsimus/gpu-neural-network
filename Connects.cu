#include "hip/hip_runtime.h"

typedef struct Connects
{
    float* widths;
    float* biases;
} Connects;

__device__ void Forward(Connects* LayerConnect, Inputs* input, Inputs* output)
{

    int outputSize = output->count;
    int inputSize = input->count;
    // float *widths = LayerConnect.widths;
    float* biases = LayerConnect->biases;
    // printf("outputSize: %d\n", outputSize);
    // printf("inputSize: %d\n", inputSize);

    for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
    {
      //  printf("Aa: %d\n", outputIndex);
        float outputElement = 0;

        for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
        {
            float inputElement = input->allocatedInputs[inputIndex];
            outputElement += inputElement * (outputIndex * inputSize + inputIndex);
        };
        (*output).allocatedInputs[outputIndex] = outputElement + biases[outputIndex];
    };
}


__device__ Connects* CreateConnection(int inputSize, int outputSize)
{
    float* widths;

    hipMalloc(&widths, inputSize * outputSize * sizeof(float));

    float* biases;

    hipMalloc(&biases, outputSize * sizeof(float));

    Connects* devicePtr;
    hipMalloc(&devicePtr, sizeof(Connects));

    (*devicePtr).widths = widths;
    (*devicePtr).biases = biases;

    return devicePtr;
};


__device__ Connects* NewGpuAllocateConnects(int size)
{
    Connects* connect;
    hipMalloc(&connect, size * sizeof(Connects));
    return connect;
}