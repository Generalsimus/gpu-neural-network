#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <corecrt_math.h>

typedef struct Connects
{
    float* widths;
    float* biases;

    dim3 blocksPerGrid;
    dim3 threadsPerBlock;

} Connects;


__global__ void ForwardSum(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths) {
     size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
     size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;
     size_t widthIndex = inputIndex * *outputsSize + outputIndex;

      

     outputs[outputIndex] += inputs[inputIndex] * widths[widthIndex];

    // printf("widthIndex = %d\n", inputIndex * *outputsSize + outputIndex);
    // widths[inputIndex * *outputsSize + outputIndex]
    // printf("Thread %d: Width value = %.2f\n", inputIndex * *outputsSize + outputIndex, widths[inputIndex * *outputsSize + outputIndex]);
    // printf("Thread %d: outputIndex value = %.2f\n", outputIndex, outputs[outputIndex]);
    // printf("Thread %d: inputIndex value = %.2f\n", inputIndex, inputs[inputIndex]);
    // printf("Thread %d: Input value = %.2f\n", inputIndex, outputs[outputIndex]);



}

__global__ void ForwardSigmoid(float* inputs, size_t* inputsSize, float* outputs, size_t* outputsSize, float* widths) {
    size_t outputIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //size_t inputIndex = blockIdx.x * blockDim.x + threadIdx.x;/*
    //size_t widthIndex = inputIndex * *outputsSize + outputIndex;*/

    outputs[outputIndex] = 1.0f / (1.0f + expf(-outputs[outputIndex]));
}
//__device__ void Forward(Connects* LayerConnect, Inputs* input, Inputs* output)
//{
//
//    int outputSize = output->count;
//    int inputSize = input->count;
//
//    float* biases = LayerConnect->biases;
//    float* widths = LayerConnect->widths;
//
//
//    for (int outputIndex = 0; outputIndex < outputSize; outputIndex++)
//    {
//
//        float outputElement = 0;
//
//        for (int inputIndex = 0; inputIndex < inputSize; inputIndex++)
//        {
//            float inputElement = input->allocatedInputs[inputIndex];
//
//            int widthIndex = ((inputIndex * outputIndex) + outputIndex);
//
//            outputElement += inputElement * widths[widthIndex];
//        };
//        (*output).allocatedInputs[outputIndex] = outputElement + biases[outputIndex];
//    };
//}
// ((inputIndex* OutputIndex) + OutputIndex)

Connects NewConnection(size_t inputSize, size_t outputSize)
{
    /////////////////////////////////////////////
    size_t inputThredBalance = FindBalanceThread(inputSize);
    size_t outputThredBalance = FindBalanceThread(outputSize);


    dim3 blocksPerGrid(inputSize / inputThredBalance, outputSize / outputThredBalance);
    dim3 threadsPerBlock(inputThredBalance, outputThredBalance);
    /////////////////////////////////////////////

    float* widths;
    hipMalloc((void**)&widths, inputSize * outputSize * sizeof(float));

    //////////////////////////////////////////////

    float* biases;
    hipMalloc((void**)&biases, outputSize * sizeof(float));

    //////////////////////////////////////////////

   
    Connects connects = {
        widths,
        biases,
        blocksPerGrid,
        threadsPerBlock,
    };
     
    return connects;
};


